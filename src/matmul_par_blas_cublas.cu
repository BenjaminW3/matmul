//-----------------------------------------------------------------------------
//! Copyright (c) 2014-2015, Benjamin Worpitz
//! All rights reserved.
//! 
//! Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met :
//! * Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
//! * Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
//! * Neither the name of the TU Dresden nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
//! 
//! THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
//! IN NO EVENT SHALL THE COPYRIGHT HOLDER BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) 
//! HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//-----------------------------------------------------------------------------

#include "matmul_par_blas_cublas.cuh"

#ifdef MATMUL_BUILD_PAR_BLAS_CUBLAS

	#include <stdio.h>		// printf

	#include <hip/hip_runtime.h>

	#define MATMUL_CUDA_CHECK(cmd) {hipError_t error = cmd; if(error!=hipSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", hipGetErrorString(error));}}

	#include <hip/hip_runtime.h>
	#include <hipblas.h>
	
	#define MATMUL_CUBLAS_CHECK(cmd) {hipblasStatus_t ret = cmd; if(ret!=HIPBLAS_STATUS_SUCCESS){printf("hipblasCreate returned error code %d, line(%d)\n", ret, __LINE__); exit(EXIT_FAILURE);}}

	//-----------------------------------------------------------------------------
	//
	//-----------------------------------------------------------------------------
	void matmul_par_blas_cublas2(
		size_t const n,
		TElement const * const restrict A,
		TElement const * const restrict B,
		TElement * const restrict C)
	{
		TElement *pADev, *pBDev, *pCDev;

		size_t const uiBytes = n*n*sizeof(TElement);
		
		MATMUL_CUDA_CHECK(hipMalloc((void **) &pADev, uiBytes));
		MATMUL_CUDA_CHECK(hipMemcpy(pADev, A, uiBytes, hipMemcpyHostToDevice));
		MATMUL_CUDA_CHECK(hipMalloc((void **) &pBDev, uiBytes));
		MATMUL_CUDA_CHECK(hipMemcpy(pBDev, B, uiBytes, hipMemcpyHostToDevice));
		MATMUL_CUDA_CHECK(hipMalloc((void **) &pCDev, uiBytes));

		// Initialise cublas
		hipblasHandle_t handle;
		MATMUL_CUBLAS_CHECK(hipblasCreate(&handle));
		
		// Do the calculation.
		TElement const alpha = 1;
		TElement const beta  = 1;
		//Note: cublas is column primary! So we need to transpose the order.
		#ifdef MATMUL_ELEMENT_TYPE_DOUBLE
			MATMUL_CUBLAS_CHECK(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, pBDev, n, pADev, n, &beta, pCDev, n));
		#else
			MATMUL_CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, pBDev, n, pADev, n, &beta, pCDev, n));
		#endif

		MATMUL_CUDA_CHECK(hipDeviceSynchronize());
		MATMUL_CUDA_CHECK(hipMemcpy(C, pCDev, uiBytes, hipMemcpyDeviceToHost));

		hipFree(pADev);
		hipFree(pBDev);
		hipFree(pCDev);

		MATMUL_CUBLAS_CHECK(hipblasDestroy(handle));
	}
#endif
