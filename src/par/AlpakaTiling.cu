
#include <hip/hip_runtime.h>
//-----------------------------------------------------------------------------
//! \file
//! Copyright 2013-2016 Benjamin Worpitz, Rene Widera
//!
//! This file is part of matmul.
//!
//! matmul is free software: you can redistribute it and/or modify
//! it under the terms of the GNU Lesser General Public License as published by
//! the Free Software Foundation, either version 3 of the License, or
//! (at your option) any later version.
//!
//! matmul is distributed in the hope that it will be useful,
//! but WITHOUT ANY WARRANTY; without even the implied warranty of
//! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//! GNU Lesser General Public License for more details.
//!
//! You should have received a copy of the GNU Lesser General Public License
//! along with matmul.
//! If not, see <http://www.gnu.org/licenses/>.
//-----------------------------------------------------------------------------

#if defined(MATMUL_BUILD_PAR_ALPAKA_ACC_GPU_CUDA) || defined(MATMUL_BUILD_PAR_ALPAKA_ACC_GPU_CUDA_MEMCPY)

    #include <matmul/par/AlpakaTiling.h>

    #include <matmul/par/AlpakaTiling.hpp>

    //-----------------------------------------------------------------------------
    //
    //-----------------------------------------------------------------------------
    TReturn matmul_gemm_par_alpaka_gpu_cuda_tiling(
        TSize const m, TSize const n, TSize const k,
        TElem const alpha,
        TElem const * const MATMUL_RESTRICT A, TSize const lda,
        TElem const * const MATMUL_RESTRICT B, TSize const ldb,
        TElem const beta,
        TElem * const MATMUL_RESTRICT C, TSize const ldc)
    {
        return
            matmul_gemm_par_alpaka_tiling<alpaka::acc::AccGpuCudaRt<alpaka::dim::DimInt<2u>, TSize>, GemmAlpakaTiling>(
                m, n, k,
                alpha,
                A, lda,
                B, ldb,
                beta,
                C, ldc);
    }
    //-----------------------------------------------------------------------------
    //
    //-----------------------------------------------------------------------------
    TReturn matmul_gemm_par_alpaka_gpu_cuda_memcpy_tiling(
        TSize const m, TSize const n, TSize const k,
        TElem const alpha,
        TElem const * const MATMUL_RESTRICT A, TSize const lda,
        TElem const * const MATMUL_RESTRICT B, TSize const ldb,
        TElem const beta,
        TElem * const MATMUL_RESTRICT C, TSize const ldc)
    {
        return
            matmul_gemm_par_alpaka_memcpy_tiling<alpaka::acc::AccGpuCudaRt<alpaka::dim::DimInt<2u>, TSize>, GemmAlpakaTiling>(
                m, n, k,
                alpha,
                A, lda,
                B, ldb,
                beta,
                C, ldc);
    }
#endif
