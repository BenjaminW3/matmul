
#include <hip/hip_runtime.h>
//-----------------------------------------------------------------------------
//! Copyright (c) 2014-2015, Benjamin Worpitz
//! All rights reserved.
//!
//! Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met :
//! * Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
//! * Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
//! * Neither the name of the TU Dresden nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
//!
//! THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
//! IN NO EVENT SHALL THE COPYRIGHT HOLDER BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
//! HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//-----------------------------------------------------------------------------

#ifdef MATMUL_BUILD_PAR_CUDA

    #include <matmul/par/Cuda.h>

    #include <matmul/common/Mat.h>  // matmul_mat_gemm_early_out

    #include <cuda_runtime.h>

    #include <stdio.h>              // printf
    #include <math.h>               // ceil

    #define MATMUL_CUDA_RT_CHECK(cmd) {cudaError_t error = cmd; if(error!=cudaSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", cudaGetErrorString(error));}}

    //-----------------------------------------------------------------------------
    // This function only works for square blocks.
    //-----------------------------------------------------------------------------
    __global__ void matmul_gemm_par_cuda_kernel(
        size_t const m, size_t const n, size_t const k,
        TElem const alpha,
        TElem const * const A, size_t const lda,
        TElem const * const B, size_t const ldb,
        TElem const beta,
        TElem * const C, size_t const ldc)
    {
        // blockIdx.x and blockIdx.y are the indices of the block to calculate inside C.
        size_t const uiGridThreadIdxX = blockIdx.x*blockDim.x + threadIdx.x;    // Column inside C to calculate.
        size_t const uiGridThreadIdxY = blockIdx.y*blockDim.y + threadIdx.y;    // Row inside C to calculate.

        size_t const uiBlockThreadIdxX = threadIdx.x;    // Column inside the block of C to calculate.
        size_t const uiBlockThreadIdxY = threadIdx.y;    // Row inside the block of C to calculate.

        size_t const uiBlockThreadsExtentX = blockDim.x;
        size_t const uiBlockThreadsExtentY = blockDim.y;
        //assert(uiBlockThreadsExtentX == uiBlockThreadsExtentY);
        size_t const uiBlockThreadsExtent = uiBlockThreadsExtentX;

        // Shared memory used to store the current blocks of A and B.
        __shared__ TElem pSharedBlockA[MATMUL_CUDA_BLOCKSIZE][MATMUL_CUDA_BLOCKSIZE];
        __shared__ TElem pSharedBlockB[MATMUL_CUDA_BLOCKSIZE][MATMUL_CUDA_BLOCKSIZE];

        // If the element is outside of the matrix, write zero into the shared block.
        bool const bInsideA = (uiGridThreadIdxY < m);
        bool const bInsideB = (uiGridThreadIdxX < n);
        bool const bInsideC = (bInsideA && bInsideB);

        TElem fCSum(0);

        // Loop over all blocks of A and B that are required to compute the C block.
        auto const uiBlockMulCount(
            static_cast<size_t>(
                ceil(
                    static_cast<float>(k)/static_cast<float>(uiBlockThreadsExtent))));
        for(size_t k2=0; k2<uiBlockMulCount; ++k2)
        {
            // Copy data to shared memory.
            auto const uiAIdxX(k2*uiBlockThreadsExtentX + uiBlockThreadIdxX);
            auto const uiAIdx1d(uiGridThreadIdxY*lda + uiAIdxX);
            pSharedBlockA[uiBlockThreadIdxY][uiBlockThreadIdxX] =
                ((!bInsideA) || (uiAIdxX>=k))
                ? static_cast<TElem>(0)
                : A[uiAIdx1d];

            auto const uiBIdxY(k2*uiBlockThreadsExtentY + uiBlockThreadIdxY);
            auto const uiBIdx1d(uiBIdxY*ldb + uiGridThreadIdxX);
            pSharedBlockB[uiBlockThreadIdxY][uiBlockThreadIdxX] =
                ((!bInsideB) || (uiBIdxY>=k))
                ? static_cast<TElem>(0)
                : B[uiBIdx1d];

            // Synchronize to make sure the sub-matrices are loaded before starting the computation.
            __syncthreads();

            // Dyadic product within shared memory.
            for(size_t k3 = 0; k3<uiBlockThreadsExtent; ++k3)
            {
                fCSum += alpha * pSharedBlockA[uiBlockThreadIdxY][k3]
                    * pSharedBlockB[k3][uiBlockThreadIdxX];
            }

            // Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration.
            __syncthreads();
        }

        if(bInsideC)
        {
            auto const uiIdxC1d(uiGridThreadIdxY*ldc + uiGridThreadIdxX);
            C[uiIdxC1d] = alpha * fCSum + beta * C[uiIdxC1d];
        }
    }
    //-----------------------------------------------------------------------------
    //
    //-----------------------------------------------------------------------------
    void matmul_gemm_par_cuda(
        size_t const m, size_t const n, size_t const k,
        TElem const alpha,
        TElem const * const MATMUL_RESTRICT A, size_t const lda,
        TElem const * const MATMUL_RESTRICT B, size_t const ldb,
        TElem const beta,
        TElem * const MATMUL_RESTRICT C, size_t const ldc)
    {
        if(matmul_mat_gemm_early_out(m, n, k, alpha, beta))
        {
            return;
        }

        cudaStream_t stream;
        MATMUL_CUDA_RT_CHECK(cudaStreamCreate(&stream));

        size_t const uiBytesA = lda*m*sizeof(TElem);
        size_t const uiBytesB = ldb*k*sizeof(TElem);
        size_t const uiBytesC = ldc*m*sizeof(TElem);

        TElem *pADev, *pBDev, *pCDev;
        MATMUL_CUDA_RT_CHECK(cudaMalloc((void **)&pADev, uiBytesA));
        MATMUL_CUDA_RT_CHECK(cudaMemcpyAsync(pADev, A, uiBytesA, cudaMemcpyHostToDevice, stream));
        MATMUL_CUDA_RT_CHECK(cudaMalloc((void **)&pBDev, uiBytesB));
        MATMUL_CUDA_RT_CHECK(cudaMemcpyAsync(pBDev, B, uiBytesB, cudaMemcpyHostToDevice, stream));
        MATMUL_CUDA_RT_CHECK(cudaMalloc((void **)&pCDev, uiBytesC));
        MATMUL_CUDA_RT_CHECK(cudaMemcpyAsync(pCDev, C, uiBytesC, cudaMemcpyHostToDevice, stream));

        dim3 const dimBlock(MATMUL_CUDA_BLOCKSIZE, MATMUL_CUDA_BLOCKSIZE);
        float const fGridThreadExtentX = ceil(((float)n)/((float)MATMUL_CUDA_BLOCKSIZE));
        float const fGridThreadExtentY = ceil(((float)m)/((float)MATMUL_CUDA_BLOCKSIZE));
        unsigned int const uiGridThreadExtentX = (unsigned int)fGridThreadExtentX;
        unsigned int const uiGridThreadExtentY = (unsigned int)fGridThreadExtentY;
        dim3 const dimGrid(uiGridThreadExtentX, uiGridThreadExtentY);

        matmul_gemm_par_cuda_kernel<<<dimGrid, dimBlock, 0, stream>>>(
            m, n, k,
            alpha,
            pADev, lda,
            pBDev, ldb,
            beta,
            pCDev, ldc);

        MATMUL_CUDA_RT_CHECK(cudaMemcpyAsync(C, pCDev, uiBytesC, cudaMemcpyDeviceToHost, stream));

        MATMUL_CUDA_RT_CHECK(cudaStreamSynchronize(stream));

        cudaFree(pADev);
        cudaFree(pBDev);
        cudaFree(pCDev);

        MATMUL_CUDA_RT_CHECK(cudaStreamDestroy(stream));
    }
#endif
