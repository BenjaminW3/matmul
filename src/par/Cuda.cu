
#include <hip/hip_runtime.h>
//-----------------------------------------------------------------------------
//! \file
//! Copyright 2013-2015 Benjamin Worpitz
//!
//! This file is part of matmul.
//!
//! matmul is free software: you can redistribute it and/or modify
//! it under the terms of the GNU Lesser General Public License as published by
//! the Free Software Foundation, either version 3 of the License, or
//! (at your option) any later version.
//!
//! matmul is distributed in the hope that it will be useful,
//! but WITHOUT ANY WARRANTY; without even the implied warranty of
//! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//! GNU Lesser General Public License for more details.
//!
//! You should have received a copy of the GNU Lesser General Public License
//! along with matmul.
//! If not, see <http://www.gnu.org/licenses/>.
//-----------------------------------------------------------------------------

#if defined(MATMUL_BUILD_PAR_CUDA_FIXED_BLOCK_SIZE) || defined(MATMUL_BUILD_PAR_CUDA_MEMCPY_FIXED_BLOCK_SIZE) || defined(MATMUL_BUILD_PAR_CUDA_DYN_BLOCK_SIZE) || defined(MATMUL_BUILD_PAR_CUDA_MEMCPY_DYN_BLOCK_SIZE)

    #include <matmul/par/Cuda.h>

    #include <matmul/common/Cuda.h> // matmul_gemm_wrap_memcpy_host_cuda_2d
    #include <matmul/common/Mat.h>  // matmul_mat_gemm_early_out

    #include <cuda_runtime.h>

    #include <stdio.h>              // printf
    #include <math.h>               // ceil
    #include <algorithm>            // std::min

    #define MATMUL_CUDA_RT_CHECK(cmd) {cudaError_t error = cmd; if(error!=cudaSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", cudaGetErrorString(error));}}

    #ifdef MATMUL_BUILD_PAR_CUDA_FIXED_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        // This function only works for square blocks.
        //-----------------------------------------------------------------------------
        __global__ void matmul_gemm_par_cuda_fixed_block_size_2d_static_shared_kernel(
            TSize const m, TSize const n, TSize const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TSize const lda,
            TElem const * const MATMUL_RESTRICT B, TSize const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TSize const ldc)
        {
            // Column and row of C to calculate.
            TSize const gridThreadIdxX = blockIdx.x*blockDim.x + threadIdx.x;
            TSize const gridThreadIdxY = blockIdx.y*blockDim.y + threadIdx.y;

            // Column and row inside the block of C to calculate.
            TSize const blockThreadIdxX = threadIdx.x;
            TSize const blockThreadIdxY = threadIdx.y;

            // The block threads extents.
            TSize const blockThreadExtentX = blockDim.x;
            TSize const blockThreadExtentY = blockDim.y;
            //assert(blockThreadExtentX == blockThreadExtentY);
            TSize const & blockThreadExtent = blockThreadExtentX;

            // Shared memory used to store the current blocks of A and B.
            __shared__ TElem pBlockSharedA[MATMUL_CUDA_FIXED_BLOCK_SIZE][MATMUL_CUDA_FIXED_BLOCK_SIZE];
            __shared__ TElem pBlockSharedB[MATMUL_CUDA_FIXED_BLOCK_SIZE][MATMUL_CUDA_FIXED_BLOCK_SIZE];

            // If the element corresponding to the current thread is outside of the respective matrix.
            bool const insideA = (gridThreadIdxY < m);
            bool const insideB = (gridThreadIdxX < n);
            bool const insideC = (insideA && insideB);

            TElem dotProduct(0);

            // Loop over all blocks of A and B that are required to compute the C block.
            TSize const blockMulCount(
                static_cast<TSize>(
                    ceil(
                        static_cast<float>(k)/static_cast<float>(blockThreadExtent))));
            for(TSize k2=0; k2<blockMulCount; ++k2)
            {
                // Copy the current blocks of A and B into shared memory in parallel.
                // If the element of the current thread is outside of the matrix, zero is written into the shared memory.
                // This is possible because zero is a result neutral extension of the matrices regarding the dot product.
                TSize const AIdxX(k2*blockThreadExtentX + blockThreadIdxX);
                TSize const AIdx1d(gridThreadIdxY*lda + AIdxX);
                pBlockSharedA[blockThreadIdxY][blockThreadIdxX] =
                    ((!insideA) || (AIdxX>=k))
                    ? static_cast<TElem>(0)
                    : A[AIdx1d];

                TSize const BIdxY(k2*blockThreadExtentY + blockThreadIdxY);
                TSize const BIdx1d(BIdxY*ldb + gridThreadIdxX);
                pBlockSharedB[blockThreadIdxY][blockThreadIdxX] =
                    ((!insideB) || (BIdxY>=k))
                    ? static_cast<TElem>(0)
                    : B[BIdx1d];

                // Synchronize to make sure the complete blocks are loaded before starting the computation.
                __syncthreads();

                // Compute the dot products within shared memory.
                for(TSize k3 = 0; k3<blockThreadExtent; ++k3)
                {
                    dotProduct += pBlockSharedA[blockThreadIdxY][k3]
                        * pBlockSharedB[k3][blockThreadIdxX];
                }

                // Synchronize to make sure that the preceding computation is done before loading the next blocks of A and B.
                __syncthreads();
            }

            if(insideC)
            {
                TSize const CIdx1d(gridThreadIdxY*ldc + gridThreadIdxX);
                C[CIdx1d] = alpha * dotProduct + beta * C[CIdx1d];
            }
        }
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        TReturn matmul_gemm_par_cuda_fixed_block_size_2d_static_shared(
            TSize const m, TSize const n, TSize const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TSize const lda,
            TElem const * const MATMUL_RESTRICT B, TSize const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TSize const ldc)
        {
            if(matmul_mat_gemm_early_out(m, n, k, alpha, beta))
            {
                MATMUL_TIME_RETURN_EARLY_OUT;
            }

            dim3 const dimBlock(MATMUL_CUDA_FIXED_BLOCK_SIZE, MATMUL_CUDA_FIXED_BLOCK_SIZE);
            float const fGridThreadExtentX = ceil(((float)n) / ((float)MATMUL_CUDA_FIXED_BLOCK_SIZE));
            float const fGridThreadExtentY = ceil(((float)m) / ((float)MATMUL_CUDA_FIXED_BLOCK_SIZE));
            unsigned int const gridThreadExtentX = (unsigned int)fGridThreadExtentX;
            unsigned int const gridThreadExtentY = (unsigned int)fGridThreadExtentY;
            dim3 const dimGrid(gridThreadExtentX, gridThreadExtentY);

            MATMUL_TIME_START;

            matmul_gemm_par_cuda_fixed_block_size_2d_static_shared_kernel<<<
                dimGrid,
                dimBlock,
                0>>>(
                    m, n, k,
                    alpha,
                    A, lda,
                    B, ldb,
                    beta,
                    C, ldc);

            MATMUL_CUDA_RT_CHECK(cudaDeviceSynchronize());
            
            MATMUL_TIME_END;
            MATMUL_TIME_RETURN;
        }
    #endif
    #ifdef MATMUL_BUILD_PAR_CUDA_MEMCPY_FIXED_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        TReturn matmul_gemm_par_cuda_memcpy_fixed_block_size_2d_static_shared(
            TSize const m, TSize const n, TSize const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TSize const lda,
            TElem const * const MATMUL_RESTRICT B, TSize const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TSize const ldc)
        {
            return
                matmul_gemm_wrap_memcpy_host_cuda_2d(
                    m, n, k,
                    alpha,
                    A, lda,
                    B, ldb,
                    beta,
                    C, ldc,
                    matmul_gemm_par_cuda_fixed_block_size_2d_static_shared);
        }
    #endif
    #ifdef MATMUL_BUILD_PAR_CUDA_FIXED_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        // This function only works for square blocks.
        //-----------------------------------------------------------------------------
        __global__ void matmul_gemm_par_cuda_fixed_block_size_1d_static_shared_kernel(
            TSize const m, TSize const n, TSize const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TSize const lda,
            TElem const * const MATMUL_RESTRICT B, TSize const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TSize const ldc)
        {
            // Column and row of C to calculate.
            TSize const gridThreadIdxX = blockIdx.x*blockDim.x + threadIdx.x;
            TSize const gridThreadIdxY = blockIdx.y*blockDim.y + threadIdx.y;

            // Column and row inside the block of C to calculate.
            TSize const blockThreadIdxX = threadIdx.x;
            TSize const blockThreadIdxY = threadIdx.y;

            // The block threads extents.
            TSize const blockThreadExtentX = blockDim.x;
            TSize const blockThreadExtentY = blockDim.y;
            //assert(blockThreadExtentX == blockThreadExtentY);
            TSize const & blockThreadExtent = blockThreadExtentX;

            // Shared memory used to store the current blocks of A and B.
            __shared__ TElem pBlockSharedA[MATMUL_CUDA_FIXED_BLOCK_SIZE*MATMUL_CUDA_FIXED_BLOCK_SIZE];
            __shared__ TElem pBlockSharedB[MATMUL_CUDA_FIXED_BLOCK_SIZE*MATMUL_CUDA_FIXED_BLOCK_SIZE];

            TSize const sharedBlockIdx1d(blockThreadIdxY*blockThreadExtentX + blockThreadIdxX);

            // If the element corresponding to the current thread is outside of the respective matrix.
            bool const insideA = (gridThreadIdxY < m);
            bool const insideB = (gridThreadIdxX < n);
            bool const insideC = (insideA && insideB);

            TElem dotProduct(0);

            // Loop over all blocks of A and B that are required to compute the C block.
            TSize const blockMulCount(
                static_cast<TSize>(
                    ceil(
                        static_cast<float>(k)/static_cast<float>(blockThreadExtent))));
            for(TSize k2=0; k2<blockMulCount; ++k2)
            {
                // Copy the current blocks of A and B into shared memory in parallel.
                // If the element of the current thread is outside of the matrix, zero is written into the shared memory.
                // This is possible because zero is a result neutral extension of the matrices regarding the dot product.
                TSize const AIdxX(k2*blockThreadExtentX + blockThreadIdxX);
                TSize const AIdx1d(gridThreadIdxY*lda + AIdxX);
                pBlockSharedA[sharedBlockIdx1d] =
                    ((!insideA) || (AIdxX>=k))
                    ? static_cast<TElem>(0)
                    : A[AIdx1d];

                TSize const BIdxY(k2*blockThreadExtentY + blockThreadIdxY);
                TSize const BIdx1d(BIdxY*ldb + gridThreadIdxX);
                pBlockSharedB[sharedBlockIdx1d] =
                    ((!insideB) || (BIdxY>=k))
                    ? static_cast<TElem>(0)
                    : B[BIdx1d];

                // Synchronize to make sure the complete blocks are loaded before starting the computation.
                __syncthreads();

                // Compute the dot products within shared memory.
                for(TSize k3 = 0; k3<blockThreadExtent; ++k3)
                {
                    dotProduct += pBlockSharedA[blockThreadIdxY*blockThreadExtentX + k3]
                        * pBlockSharedB[k3*blockThreadExtentY + blockThreadIdxX];
                }

                // Synchronize to make sure that the preceding computation is done before loading the next blocks of A and B.
                __syncthreads();
            }

            if(insideC)
            {
                TSize const CIdx1d(gridThreadIdxY*ldc + gridThreadIdxX);
                C[CIdx1d] = alpha * dotProduct + beta * C[CIdx1d];
            }
        }
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        TReturn matmul_gemm_par_cuda_fixed_block_size_1d_static_shared(
            TSize const m, TSize const n, TSize const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TSize const lda,
            TElem const * const MATMUL_RESTRICT B, TSize const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TSize const ldc)
        {
            if(matmul_mat_gemm_early_out(m, n, k, alpha, beta))
            {
                MATMUL_TIME_RETURN_EARLY_OUT;
            }

            dim3 const dimBlock(MATMUL_CUDA_FIXED_BLOCK_SIZE, MATMUL_CUDA_FIXED_BLOCK_SIZE);
            float const fGridThreadExtentX = ceil(((float)n) / ((float)MATMUL_CUDA_FIXED_BLOCK_SIZE));
            float const fGridThreadExtentY = ceil(((float)m) / ((float)MATMUL_CUDA_FIXED_BLOCK_SIZE));
            unsigned int const gridThreadExtentX = (unsigned int)fGridThreadExtentX;
            unsigned int const gridThreadExtentY = (unsigned int)fGridThreadExtentY;
            dim3 const dimGrid(gridThreadExtentX, gridThreadExtentY);

            MATMUL_TIME_START;

            matmul_gemm_par_cuda_fixed_block_size_1d_static_shared_kernel<<<
                dimGrid,
                dimBlock,
                0>>>(
                    m, n, k,
                    alpha,
                    A, lda,
                    B, ldb,
                    beta,
                    C, ldc);

            MATMUL_CUDA_RT_CHECK(cudaDeviceSynchronize());
            
            MATMUL_TIME_END;
            MATMUL_TIME_RETURN;
        }
    #endif
    #ifdef MATMUL_BUILD_PAR_CUDA_MEMCPY_FIXED_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        TReturn matmul_gemm_par_cuda_memcpy_fixed_block_size_1d_static_shared(
            TSize const m, TSize const n, TSize const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TSize const lda,
            TElem const * const MATMUL_RESTRICT B, TSize const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TSize const ldc)
        {
            return
                matmul_gemm_wrap_memcpy_host_cuda_2d(
                    m, n, k,
                    alpha,
                    A, lda,
                    B, ldb,
                    beta,
                    C, ldc,
                    matmul_gemm_par_cuda_fixed_block_size_1d_static_shared);
        }
    #endif
    #ifdef MATMUL_BUILD_PAR_CUDA_FIXED_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        // This function only works for square blocks.
        //-----------------------------------------------------------------------------
        __global__ void matmul_gemm_par_cuda_fixed_block_size_1d_extern_shared_kernel(
            TSize const m, TSize const n, TSize const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TSize const lda,
            TElem const * const MATMUL_RESTRICT B, TSize const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TSize const ldc)
        {
            // Column and row of C to calculate.
            TSize const gridThreadIdxX = blockIdx.x*blockDim.x + threadIdx.x;
            TSize const gridThreadIdxY = blockIdx.y*blockDim.y + threadIdx.y;

            // Column and row inside the block of C to calculate.
            TSize const blockThreadIdxX = threadIdx.x;
            TSize const blockThreadIdxY = threadIdx.y;

            // The block threads extents.
            TSize const blockThreadExtentX = blockDim.x;
            TSize const blockThreadExtentY = blockDim.y;
            //assert(blockThreadExtentX == blockThreadExtentY);
            TSize const & blockThreadExtent = blockThreadExtentX;

            // Shared memory used to store the current blocks of A and B.
            extern __shared__ TElem pBlockSharedA[];
            TElem * const pBlockSharedB(pBlockSharedA + blockThreadExtentX*blockThreadExtentY);

            TSize const sharedBlockIdx1d(blockThreadIdxY*blockThreadExtentX + blockThreadIdxX);

            // If the element corresponding to the current thread is outside of the respective matrix.
            bool const insideA = (gridThreadIdxY < m);
            bool const insideB = (gridThreadIdxX < n);
            bool const insideC = (insideA && insideB);

            TElem dotProduct(0);

            // Loop over all blocks of A and B that are required to compute the C block.
            TSize const blockMulCount(
                static_cast<TSize>(
                    ceil(
                        static_cast<float>(k)/static_cast<float>(blockThreadExtent))));
            for(TSize k2=0; k2<blockMulCount; ++k2)
            {
                // Copy the current blocks of A and B into shared memory in parallel.
                // If the element of the current thread is outside of the matrix, zero is written into the shared memory.
                // This is possible because zero is a result neutral extension of the matrices regarding the dot product.
                TSize const AIdxX(k2*blockThreadExtentX + blockThreadIdxX);
                TSize const AIdx1d(gridThreadIdxY*lda + AIdxX);
                pBlockSharedA[sharedBlockIdx1d] =
                    ((!insideA) || (AIdxX>=k))
                    ? static_cast<TElem>(0)
                    : A[AIdx1d];

                TSize const BIdxY(k2*blockThreadExtentY + blockThreadIdxY);
                TSize const BIdx1d(BIdxY*ldb + gridThreadIdxX);
                pBlockSharedB[sharedBlockIdx1d] =
                    ((!insideB) || (BIdxY>=k))
                    ? static_cast<TElem>(0)
                    : B[BIdx1d];

                // Synchronize to make sure the complete blocks are loaded before starting the computation.
                __syncthreads();

                // Compute the dot products within shared memory.
                for(TSize k3 = 0; k3<blockThreadExtent; ++k3)
                {
                    dotProduct += pBlockSharedA[blockThreadIdxY*blockThreadExtentX + k3]
                        * pBlockSharedB[k3*blockThreadExtentY + blockThreadIdxX];
                }

                // Synchronize to make sure that the preceding computation is done before loading the next blocks of A and B.
                __syncthreads();
            }

            if(insideC)
            {
                TSize const CIdx1d(gridThreadIdxY*ldc + gridThreadIdxX);
                C[CIdx1d] = alpha * dotProduct + beta * C[CIdx1d];
            }
        }
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        TReturn matmul_gemm_par_cuda_fixed_block_size_1d_extern_shared(
            TSize const m, TSize const n, TSize const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TSize const lda,
            TElem const * const MATMUL_RESTRICT B, TSize const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TSize const ldc)
        {
            if(matmul_mat_gemm_early_out(m, n, k, alpha, beta))
            {
                MATMUL_TIME_RETURN_EARLY_OUT;
            }

            dim3 const dimBlock(MATMUL_CUDA_FIXED_BLOCK_SIZE, MATMUL_CUDA_FIXED_BLOCK_SIZE);
            float const fGridThreadExtentX = ceil(((float)n) / ((float)MATMUL_CUDA_FIXED_BLOCK_SIZE));
            float const fGridThreadExtentY = ceil(((float)m) / ((float)MATMUL_CUDA_FIXED_BLOCK_SIZE));
            unsigned int const gridThreadExtentX = (unsigned int)fGridThreadExtentX;
            unsigned int const gridThreadExtentY = (unsigned int)fGridThreadExtentY;
            dim3 const dimGrid(gridThreadExtentX, gridThreadExtentY);

            MATMUL_TIME_START;

            matmul_gemm_par_cuda_fixed_block_size_1d_extern_shared_kernel<<<
                dimGrid,
                dimBlock,
                2u*sizeof(TElem)*MATMUL_CUDA_FIXED_BLOCK_SIZE*MATMUL_CUDA_FIXED_BLOCK_SIZE>>>(
                    m, n, k,
                    alpha,
                    A, lda,
                    B, ldb,
                    beta,
                    C, ldc);

            MATMUL_CUDA_RT_CHECK(cudaDeviceSynchronize());
            
            MATMUL_TIME_END;
            MATMUL_TIME_RETURN;
        }
    #endif
    #ifdef MATMUL_BUILD_PAR_CUDA_MEMCPY_FIXED_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        TReturn matmul_gemm_par_cuda_memcpy_fixed_block_size_1d_extern_shared(
            TSize const m, TSize const n, TSize const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TSize const lda,
            TElem const * const MATMUL_RESTRICT B, TSize const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TSize const ldc)
        {
            return
                matmul_gemm_wrap_memcpy_host_cuda_2d(
                    m, n, k,
                    alpha,
                    A, lda,
                    B, ldb,
                    beta,
                    C, ldc,
                    matmul_gemm_par_cuda_fixed_block_size_1d_extern_shared);
        }
    #endif
    #ifdef MATMUL_BUILD_PAR_CUDA_DYN_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        // This function only works for square blocks.
        //-----------------------------------------------------------------------------
        __global__ void matmul_gemm_par_cuda_dyn_block_size_1d_extern_shared_kernel(
            TSize const m, TSize const n, TSize const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TSize const lda,
            TElem const * const MATMUL_RESTRICT B, TSize const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TSize const ldc)
        {
            // Column and row of C to calculate.
            TSize const gridThreadIdxX = blockIdx.x*blockDim.x + threadIdx.x;
            TSize const gridThreadIdxY = blockIdx.y*blockDim.y + threadIdx.y;

            // Column and row inside the block of C to calculate.
            TSize const blockThreadIdxX = threadIdx.x;
            TSize const blockThreadIdxY = threadIdx.y;

            // The block threads extents.
            TSize const blockThreadExtentX = blockDim.x;
            TSize const blockThreadExtentY = blockDim.y;
            //assert(blockThreadExtentX == blockThreadExtentY);
            TSize const & blockThreadExtent1d = blockThreadExtentX;

            // Shared memory used to store the current blocks of A and B.
            extern __shared__ TElem pBlockSharedA[];
            TElem * const pBlockSharedB(pBlockSharedA + blockThreadExtentX*blockThreadExtentY);

            TSize const sharedBlockIdx1d(blockThreadIdxY*blockThreadExtentX + blockThreadIdxX);

            // If the element corresponding to the current thread is outside of the respective matrix.
            bool const insideA = (gridThreadIdxY < m);
            bool const insideB = (gridThreadIdxX < n);
            bool const insideC = (insideA && insideB);

            TElem dotProduct(0);

            // Loop over all blocks of A and B that are required to compute the C block.
            TSize const blockMulCount(
                static_cast<TSize>(
                    ceil(
                        static_cast<float>(k) / static_cast<float>(blockThreadExtent1d))));
            for(TSize k2(0); k2<blockMulCount; ++k2)
            {
                // Copy the current blocks of A and B into shared memory in parallel.
                // If the element of the current thread is outside of the matrix, zero is written into the shared memory.
                // This is possible because zero is a result neutral extension of the matrices regarding the dot product.
                TSize const AIdxX(k2*blockThreadExtentX + blockThreadIdxX);
                TSize const AIdx1d(gridThreadIdxY*lda + AIdxX);
                pBlockSharedA[sharedBlockIdx1d] =
                    ((!insideA) || (AIdxX >= k))
                    ? static_cast<TElem>(0)
                    : A[AIdx1d];

                TSize const BIdxY(k2*blockThreadExtentY + blockThreadIdxY);
                TSize const BIdx1d(BIdxY*ldb + gridThreadIdxX);
                pBlockSharedB[sharedBlockIdx1d] =
                    ((!insideB) || (BIdxY >= k))
                    ? static_cast<TElem>(0)
                    : B[BIdx1d];

                // Synchronize to make sure the complete blocks are loaded before starting the computation.
                __syncthreads();

                // Compute the dot products within shared memory.
                for(TSize k3(0); k3<blockThreadExtent1d; ++k3)
                {
                    dotProduct += pBlockSharedA[blockThreadIdxY*blockThreadExtentX + k3]
                        * pBlockSharedB[k3*blockThreadExtentY + blockThreadIdxX];
                }

                // Synchronize to make sure that the preceding computation is done before loading the next blocks of A and B.
                __syncthreads();
            }

            if (insideC)
            {
                TSize const CIdx1d(gridThreadIdxY*ldc + gridThreadIdxX);
                C[CIdx1d] = alpha * dotProduct + beta * C[CIdx1d];
            }
        }
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        TReturn matmul_gemm_par_cuda_dyn_block_size_1d_extern_shared(
            TSize const m, TSize const n, TSize const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TSize const lda,
            TElem const * const MATMUL_RESTRICT B, TSize const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TSize const ldc)
        {
            if(matmul_mat_gemm_early_out(m, n, k, alpha, beta))
            {
                MATMUL_TIME_RETURN_EARLY_OUT;
            }

            // MATMUL_CUDA_RT_CHECK(cudaSetDevice(0));
            cudaStream_t stream;
            MATMUL_CUDA_RT_CHECK(cudaStreamCreate(&stream));

            // Get its properties.
            cudaDeviceProp cudaDevProp;
            MATMUL_CUDA_RT_CHECK(cudaGetDeviceProperties(
                &cudaDevProp,
                0));

            TSize gridThreadExtent[] = {m, n};
            TSize blockThreadExtent[] = {cudaDevProp.maxThreadsDim[0], cudaDevProp.maxThreadsDim[1]};

            // Restrict the max block thread extent with the grid thread extent.
            // This removes dimensions not required in the given grid thread extent.
            // This has to be done before the maxThreadsPerBlock clipping to get the maximum correctly.
            for(TSize i(0); i<2; ++i)
            {
                blockThreadExtent[i] = std::min(blockThreadExtent[i], gridThreadExtent[i]);
            }

            // Restrict it to its minimum component.
            // For example (512, 256) will get (256, 256).
            TSize minBlockThreadExtent(blockThreadExtent[0]);
            for(TSize i(1); i<2; ++i)
            {
                minBlockThreadExtent = std::min(minBlockThreadExtent, blockThreadExtent[i]);
            }
            for(TSize i(0); i<2; ++i)
            {
                blockThreadExtent[i] = minBlockThreadExtent;
            }

            // Adjust blockThreadExtent if its product is too large.
            if ((blockThreadExtent[0] * blockThreadExtent[1]) > cudaDevProp.maxThreadsPerBlock)
            {
                // Satisfy the following equation:
                // udaDevProp.maxThreadsPerBlock >= blockThreadExtent[0]*blockThreadExtent[1]
                // For example 1024 >= 512 * 512

                // For equal block thread extent this is easily the nth root of cudaDevProp.maxThreadsPerBlock.
                double const fNthRoot(std::pow(cudaDevProp.maxThreadsPerBlock, 1.0 / 2.0));
                TSize const nthRoot(static_cast<TSize>(fNthRoot));
                for(TSize i(0); i<2; ++i)
                {
                    blockThreadExtent[i] = nthRoot;
                }
            }

            // Set the grid block extent (rounded to the next integer not less then the quotient.
            TSize gridBlockExtent[] = {1, 1};
            for(TSize i(0); i<2; ++i)
            {
                gridBlockExtent[i] =
                    static_cast<TSize>(
                        std::ceil(static_cast<double>(gridThreadExtent[i])
                            / static_cast<double>(blockThreadExtent[i])));
            }

            dim3 const dimBlock(blockThreadExtent[0], blockThreadExtent[1]);
            dim3 const dimGrid(gridBlockExtent[0], gridBlockExtent[1]);

            MATMUL_TIME_START;

            matmul_gemm_par_cuda_dyn_block_size_1d_extern_shared_kernel<<<
                dimGrid,
                dimBlock,
                2u*sizeof(TElem)*blockThreadExtent[0] * blockThreadExtent[1],
                stream>>>(
                    m, n, k,
                    alpha,
                    A, lda,
                    B, ldb,
                    beta,
                    C, ldc);

            MATMUL_CUDA_RT_CHECK(cudaStreamSynchronize(stream));

            MATMUL_TIME_END;

            MATMUL_CUDA_RT_CHECK(cudaStreamDestroy(stream));

            MATMUL_TIME_RETURN;
        }
    #endif
    #ifdef MATMUL_BUILD_PAR_CUDA_MEMCPY_DYN_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        TReturn matmul_gemm_par_cuda_memcpy_dyn_block_size_1d_extern_shared(
            TSize const m, TSize const n, TSize const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TSize const lda,
            TElem const * const MATMUL_RESTRICT B, TSize const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TSize const ldc)
        {
            return
                matmul_gemm_wrap_memcpy_host_cuda_2d(
                    m, n, k,
                    alpha,
                    A, lda,
                    B, ldb,
                    beta,
                    C, ldc,
                    matmul_gemm_par_cuda_dyn_block_size_1d_extern_shared);
        }
    #endif
#endif
