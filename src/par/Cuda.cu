
#include <hip/hip_runtime.h>
//-----------------------------------------------------------------------------
//! \file
//! Copyright 2013-2015 Benjamin Worpitz
//!
//! This file is part of matmul.
//!
//! matmul is free software: you can redistribute it and/or modify
//! it under the terms of the GNU Lesser General Public License as published by
//! the Free Software Foundation, either version 3 of the License, or
//! (at your option) any later version.
//!
//! matmul is distributed in the hope that it will be useful,
//! but WITHOUT ANY WARRANTY; without even the implied warranty of
//! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//! GNU Lesser General Public License for more details.
//!
//! You should have received a copy of the GNU Lesser General Public License
//! along with matmul.
//! If not, see <http://www.gnu.org/licenses/>.
//-----------------------------------------------------------------------------

#if defined(MATMUL_BUILD_PAR_CUDA_FIXED_BLOCK_SIZE) || defined(MATMUL_BUILD_PAR_CUDA_MEMCPY_FIXED_BLOCK_SIZE) || defined(MATMUL_BUILD_PAR_CUDA_DYN_BLOCK_SIZE) || defined(MATMUL_BUILD_PAR_CUDA_MEMCPY_DYN_BLOCK_SIZE)

    #include <matmul/par/Cuda.h>

    #include <matmul/common/Cuda.h> // matmul_gemm_wrap_memcpy_host_cuda_2d
    #include <matmul/common/Mat.h>  // matmul_mat_gemm_early_out

    #include <cuda_runtime.h>

    #include <stdio.h>              // printf
    #include <math.h>               // ceil
    #include <algorithm>            // std::min

    #define MATMUL_CUDA_RT_CHECK(cmd) {cudaError_t error = cmd; if(error!=cudaSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", cudaGetErrorString(error));}}

    #ifdef MATMUL_BUILD_PAR_CUDA_FIXED_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        // This function only works for square blocks.
        //-----------------------------------------------------------------------------
        __global__ void matmul_gemm_par_cuda_fixed_block_size_2d_static_shared_kernel(
            TIdx const m, TIdx const n, TIdx const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TIdx const lda,
            TElem const * const MATMUL_RESTRICT B, TIdx const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TIdx const ldc)
        {
            // Column and row of C to calculate.
            TIdx const uiGridThreadIdxX = blockIdx.x*blockDim.x + threadIdx.x;
            TIdx const uiGridThreadIdxY = blockIdx.y*blockDim.y + threadIdx.y;

            // Column and row inside the block of C to calculate.
            TIdx const uiBlockThreadIdxX = threadIdx.x;
            TIdx const uiBlockThreadIdxY = threadIdx.y;

            // The block threads extents.
            TIdx const uiBlockThreadsExtentX = blockDim.x;
            TIdx const uiBlockThreadsExtentY = blockDim.y;
            //assert(uiBlockThreadsExtentX == uiBlockThreadsExtentY);
            TIdx const & uiBlockThreadsExtent = uiBlockThreadsExtentX;

            // Shared memory used to store the current blocks of A and B.
            __shared__ TElem pBlockSharedA[MATMUL_CUDA_FIXED_BLOCK_SIZE][MATMUL_CUDA_FIXED_BLOCK_SIZE];
            __shared__ TElem pBlockSharedB[MATMUL_CUDA_FIXED_BLOCK_SIZE][MATMUL_CUDA_FIXED_BLOCK_SIZE];

            // If the element corresponding to the current thread is outside of the respective matrix.
            bool const bInsideA = (uiGridThreadIdxY < m);
            bool const bInsideB = (uiGridThreadIdxX < n);
            bool const bInsideC = (bInsideA && bInsideB);

            TElem dotProduct(0);

            // Loop over all blocks of A and B that are required to compute the C block.
            auto const uiBlockMulCount(
                static_cast<TIdx>(
                    ceil(
                        static_cast<float>(k)/static_cast<float>(uiBlockThreadsExtent))));
            for(TIdx k2=0; k2<uiBlockMulCount; ++k2)
            {
                // Copy the current blocks of A and B into shared memory in parallel.
                // If the element of the current thread is outside of the matrix, zero is written into the shared memory.
                // This is possible because zero is a result neutral extension of the matrices regarding the dot product.
                TIdx const uiAIdxX(k2*uiBlockThreadsExtentX + uiBlockThreadIdxX);
                TIdx const uiAIdx1d(uiGridThreadIdxY*lda + uiAIdxX);
                pBlockSharedA[uiBlockThreadIdxY][uiBlockThreadIdxX] =
                    ((!bInsideA) || (uiAIdxX>=k))
                    ? static_cast<TElem>(0)
                    : A[uiAIdx1d];

                TIdx const uiBIdxY(k2*uiBlockThreadsExtentY + uiBlockThreadIdxY);
                TIdx const uiBIdx1d(uiBIdxY*ldb + uiGridThreadIdxX);
                pBlockSharedB[uiBlockThreadIdxY][uiBlockThreadIdxX] =
                    ((!bInsideB) || (uiBIdxY>=k))
                    ? static_cast<TElem>(0)
                    : B[uiBIdx1d];

                // Synchronize to make sure the complete blocks are loaded before starting the computation.
                __syncthreads();

                // Compute the dot products within shared memory.
                for(TIdx k3 = 0; k3<uiBlockThreadsExtent; ++k3)
                {
                    dotProduct += pBlockSharedA[uiBlockThreadIdxY][k3]
                        * pBlockSharedB[k3][uiBlockThreadIdxX];
                }

                // Synchronize to make sure that the preceding computation is done before loading the next blocks of A and B.
                __syncthreads();
            }

            if(bInsideC)
            {
                auto const uiIdxC1d(uiGridThreadIdxY*ldc + uiGridThreadIdxX);
                C[uiIdxC1d] = alpha * dotProduct + beta * C[uiIdxC1d];
            }
        }
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        void matmul_gemm_par_cuda_fixed_block_size_2d_static_shared(
            TIdx const m, TIdx const n, TIdx const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TIdx const lda,
            TElem const * const MATMUL_RESTRICT B, TIdx const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TIdx const ldc)
        {
            if(matmul_mat_gemm_early_out(m, n, k, alpha, beta))
            {
                return;
            }

            dim3 const dimBlock(MATMUL_CUDA_FIXED_BLOCK_SIZE, MATMUL_CUDA_FIXED_BLOCK_SIZE);
            float const fGridThreadExtentX = ceil(((float)n) / ((float)MATMUL_CUDA_FIXED_BLOCK_SIZE));
            float const fGridThreadExtentY = ceil(((float)m) / ((float)MATMUL_CUDA_FIXED_BLOCK_SIZE));
            unsigned int const uiGridThreadExtentX = (unsigned int)fGridThreadExtentX;
            unsigned int const uiGridThreadExtentY = (unsigned int)fGridThreadExtentY;
            dim3 const dimGrid(uiGridThreadExtentX, uiGridThreadExtentY);

            matmul_gemm_par_cuda_fixed_block_size_2d_static_shared_kernel<<<
                dimGrid,
                dimBlock,
                0>>>(
                    m, n, k,
                    alpha,
                    A, lda,
                    B, ldb,
                    beta,
                    C, ldc);

            MATMUL_CUDA_RT_CHECK(cudaDeviceSynchronize());
        }
    #endif
    #ifdef MATMUL_BUILD_PAR_CUDA_MEMCPY_FIXED_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        void matmul_gemm_par_cuda_memcpy_fixed_block_size_2d_static_shared(
            TIdx const m, TIdx const n, TIdx const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TIdx const lda,
            TElem const * const MATMUL_RESTRICT B, TIdx const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TIdx const ldc)
        {
            matmul_gemm_wrap_memcpy_host_cuda_2d(
                m, n, k,
                alpha,
                A, lda,
                B, ldb,
                beta,
                C, ldc,
                matmul_gemm_par_cuda_fixed_block_size_2d_static_shared);
        }
    #endif
    #ifdef MATMUL_BUILD_PAR_CUDA_FIXED_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        // This function only works for square blocks.
        //-----------------------------------------------------------------------------
        __global__ void matmul_gemm_par_cuda_fixed_block_size_1d_static_shared_kernel(
            TIdx const m, TIdx const n, TIdx const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TIdx const lda,
            TElem const * const MATMUL_RESTRICT B, TIdx const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TIdx const ldc)
        {
            // Column and row of C to calculate.
            TIdx const uiGridThreadIdxX = blockIdx.x*blockDim.x + threadIdx.x;
            TIdx const uiGridThreadIdxY = blockIdx.y*blockDim.y + threadIdx.y;

            // Column and row inside the block of C to calculate.
            TIdx const uiBlockThreadIdxX = threadIdx.x;
            TIdx const uiBlockThreadIdxY = threadIdx.y;

            // The block threads extents.
            TIdx const uiBlockThreadsExtentX = blockDim.x;
            TIdx const uiBlockThreadsExtentY = blockDim.y;
            //assert(uiBlockThreadsExtentX == uiBlockThreadsExtentY);
            TIdx const & uiBlockThreadsExtent = uiBlockThreadsExtentX;

            // Shared memory used to store the current blocks of A and B.
            __shared__ TElem pBlockSharedA[MATMUL_CUDA_FIXED_BLOCK_SIZE*MATMUL_CUDA_FIXED_BLOCK_SIZE];
            __shared__ TElem pBlockSharedB[MATMUL_CUDA_FIXED_BLOCK_SIZE*MATMUL_CUDA_FIXED_BLOCK_SIZE];

            auto const uiSharedBlockIdx1d(uiBlockThreadIdxY*uiBlockThreadsExtentX + uiBlockThreadIdxX);

            // If the element corresponding to the current thread is outside of the respective matrix.
            bool const bInsideA = (uiGridThreadIdxY < m);
            bool const bInsideB = (uiGridThreadIdxX < n);
            bool const bInsideC = (bInsideA && bInsideB);

            TElem dotProduct(0);

            // Loop over all blocks of A and B that are required to compute the C block.
            auto const uiBlockMulCount(
                static_cast<TIdx>(
                    ceil(
                        static_cast<float>(k)/static_cast<float>(uiBlockThreadsExtent))));
            for(TIdx k2=0; k2<uiBlockMulCount; ++k2)
            {
                // Copy the current blocks of A and B into shared memory in parallel.
                // If the element of the current thread is outside of the matrix, zero is written into the shared memory.
                // This is possible because zero is a result neutral extension of the matrices regarding the dot product.
                TIdx const uiAIdxX(k2*uiBlockThreadsExtentX + uiBlockThreadIdxX);
                TIdx const uiAIdx1d(uiGridThreadIdxY*lda + uiAIdxX);
                pBlockSharedA[uiSharedBlockIdx1d] =
                    ((!bInsideA) || (uiAIdxX>=k))
                    ? static_cast<TElem>(0)
                    : A[uiAIdx1d];

                TIdx const uiBIdxY(k2*uiBlockThreadsExtentY + uiBlockThreadIdxY);
                TIdx const uiBIdx1d(uiBIdxY*ldb + uiGridThreadIdxX);
                pBlockSharedB[uiSharedBlockIdx1d] =
                    ((!bInsideB) || (uiBIdxY>=k))
                    ? static_cast<TElem>(0)
                    : B[uiBIdx1d];

                // Synchronize to make sure the complete blocks are loaded before starting the computation.
                __syncthreads();

                // Compute the dot products within shared memory.
                for(TIdx k3 = 0; k3<uiBlockThreadsExtent; ++k3)
                {
                    dotProduct += pBlockSharedA[uiBlockThreadIdxY*uiBlockThreadsExtentX + k3]
                        * pBlockSharedB[k3*uiBlockThreadsExtentY + uiBlockThreadIdxX];
                }

                // Synchronize to make sure that the preceding computation is done before loading the next blocks of A and B.
                __syncthreads();
            }

            if(bInsideC)
            {
                auto const uiIdxC1d(uiGridThreadIdxY*ldc + uiGridThreadIdxX);
                C[uiIdxC1d] = alpha * dotProduct + beta * C[uiIdxC1d];
            }
        }
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        void matmul_gemm_par_cuda_fixed_block_size_1d_static_shared(
            TIdx const m, TIdx const n, TIdx const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TIdx const lda,
            TElem const * const MATMUL_RESTRICT B, TIdx const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TIdx const ldc)
        {
            if(matmul_mat_gemm_early_out(m, n, k, alpha, beta))
            {
                return;
            }

            dim3 const dimBlock(MATMUL_CUDA_FIXED_BLOCK_SIZE, MATMUL_CUDA_FIXED_BLOCK_SIZE);
            float const fGridThreadExtentX = ceil(((float)n) / ((float)MATMUL_CUDA_FIXED_BLOCK_SIZE));
            float const fGridThreadExtentY = ceil(((float)m) / ((float)MATMUL_CUDA_FIXED_BLOCK_SIZE));
            unsigned int const uiGridThreadExtentX = (unsigned int)fGridThreadExtentX;
            unsigned int const uiGridThreadExtentY = (unsigned int)fGridThreadExtentY;
            dim3 const dimGrid(uiGridThreadExtentX, uiGridThreadExtentY);

            matmul_gemm_par_cuda_fixed_block_size_1d_static_shared_kernel<<<
                dimGrid,
                dimBlock,
                0>>>(
                    m, n, k,
                    alpha,
                    A, lda,
                    B, ldb,
                    beta,
                    C, ldc);

            MATMUL_CUDA_RT_CHECK(cudaDeviceSynchronize());
        }
    #endif
    #ifdef MATMUL_BUILD_PAR_CUDA_MEMCPY_FIXED_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        void matmul_gemm_par_cuda_memcpy_fixed_block_size_1d_static_shared(
            TIdx const m, TIdx const n, TIdx const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TIdx const lda,
            TElem const * const MATMUL_RESTRICT B, TIdx const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TIdx const ldc)
        {
            matmul_gemm_wrap_memcpy_host_cuda_2d(
                m, n, k,
                alpha,
                A, lda,
                B, ldb,
                beta,
                C, ldc,
                matmul_gemm_par_cuda_fixed_block_size_1d_static_shared);
        }
    #endif
    #ifdef MATMUL_BUILD_PAR_CUDA_FIXED_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        // This function only works for square blocks.
        //-----------------------------------------------------------------------------
        __global__ void matmul_gemm_par_cuda_fixed_block_size_1d_extern_shared_kernel(
            TIdx const m, TIdx const n, TIdx const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TIdx const lda,
            TElem const * const MATMUL_RESTRICT B, TIdx const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TIdx const ldc)
        {
            // Column and row of C to calculate.
            TIdx const uiGridThreadIdxX = blockIdx.x*blockDim.x + threadIdx.x;
            TIdx const uiGridThreadIdxY = blockIdx.y*blockDim.y + threadIdx.y;

            // Column and row inside the block of C to calculate.
            TIdx const uiBlockThreadIdxX = threadIdx.x;
            TIdx const uiBlockThreadIdxY = threadIdx.y;

            // The block threads extents.
            TIdx const uiBlockThreadsExtentX = blockDim.x;
            TIdx const uiBlockThreadsExtentY = blockDim.y;
            //assert(uiBlockThreadsExtentX == uiBlockThreadsExtentY);
            TIdx const & uiBlockThreadsExtent = uiBlockThreadsExtentX;

            // Shared memory used to store the current blocks of A and B.
            extern __shared__ TElem pBlockSharedA[];
            auto * const pBlockSharedB(pBlockSharedA + uiBlockThreadsExtentX*uiBlockThreadsExtentY);

            auto const uiSharedBlockIdx1d(uiBlockThreadIdxY*uiBlockThreadsExtentX + uiBlockThreadIdxX);

            // If the element corresponding to the current thread is outside of the respective matrix.
            bool const bInsideA = (uiGridThreadIdxY < m);
            bool const bInsideB = (uiGridThreadIdxX < n);
            bool const bInsideC = (bInsideA && bInsideB);

            TElem dotProduct(0);

            // Loop over all blocks of A and B that are required to compute the C block.
            auto const uiBlockMulCount(
                static_cast<TIdx>(
                    ceil(
                        static_cast<float>(k)/static_cast<float>(uiBlockThreadsExtent))));
            for(TIdx k2=0; k2<uiBlockMulCount; ++k2)
            {
                // Copy the current blocks of A and B into shared memory in parallel.
                // If the element of the current thread is outside of the matrix, zero is written into the shared memory.
                // This is possible because zero is a result neutral extension of the matrices regarding the dot product.
                TIdx const uiAIdxX(k2*uiBlockThreadsExtentX + uiBlockThreadIdxX);
                TIdx const uiAIdx1d(uiGridThreadIdxY*lda + uiAIdxX);
                pBlockSharedA[uiSharedBlockIdx1d] =
                    ((!bInsideA) || (uiAIdxX>=k))
                    ? static_cast<TElem>(0)
                    : A[uiAIdx1d];

                TIdx const uiBIdxY(k2*uiBlockThreadsExtentY + uiBlockThreadIdxY);
                TIdx const uiBIdx1d(uiBIdxY*ldb + uiGridThreadIdxX);
                pBlockSharedB[uiSharedBlockIdx1d] =
                    ((!bInsideB) || (uiBIdxY>=k))
                    ? static_cast<TElem>(0)
                    : B[uiBIdx1d];

                // Synchronize to make sure the complete blocks are loaded before starting the computation.
                __syncthreads();

                // Compute the dot products within shared memory.
                for(TIdx k3 = 0; k3<uiBlockThreadsExtent; ++k3)
                {
                    dotProduct += pBlockSharedA[uiBlockThreadIdxY*uiBlockThreadsExtentX + k3]
                        * pBlockSharedB[k3*uiBlockThreadsExtentY + uiBlockThreadIdxX];
                }

                // Synchronize to make sure that the preceding computation is done before loading the next blocks of A and B.
                __syncthreads();
            }

            if(bInsideC)
            {
                auto const uiIdxC1d(uiGridThreadIdxY*ldc + uiGridThreadIdxX);
                C[uiIdxC1d] = alpha * dotProduct + beta * C[uiIdxC1d];
            }
        }
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        void matmul_gemm_par_cuda_fixed_block_size_1d_extern_shared(
            TIdx const m, TIdx const n, TIdx const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TIdx const lda,
            TElem const * const MATMUL_RESTRICT B, TIdx const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TIdx const ldc)
        {
            if(matmul_mat_gemm_early_out(m, n, k, alpha, beta))
            {
                return;
            }

            dim3 const dimBlock(MATMUL_CUDA_FIXED_BLOCK_SIZE, MATMUL_CUDA_FIXED_BLOCK_SIZE);
            float const fGridThreadExtentX = ceil(((float)n) / ((float)MATMUL_CUDA_FIXED_BLOCK_SIZE));
            float const fGridThreadExtentY = ceil(((float)m) / ((float)MATMUL_CUDA_FIXED_BLOCK_SIZE));
            unsigned int const uiGridThreadExtentX = (unsigned int)fGridThreadExtentX;
            unsigned int const uiGridThreadExtentY = (unsigned int)fGridThreadExtentY;
            dim3 const dimGrid(uiGridThreadExtentX, uiGridThreadExtentY);

            matmul_gemm_par_cuda_fixed_block_size_1d_extern_shared_kernel<<<
                dimGrid,
                dimBlock,
                2u*sizeof(TElem)*MATMUL_CUDA_FIXED_BLOCK_SIZE*MATMUL_CUDA_FIXED_BLOCK_SIZE>>>(
                    m, n, k,
                    alpha,
                    A, lda,
                    B, ldb,
                    beta,
                    C, ldc);

            MATMUL_CUDA_RT_CHECK(cudaDeviceSynchronize());
        }
    #endif
    #ifdef MATMUL_BUILD_PAR_CUDA_MEMCPY_FIXED_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        void matmul_gemm_par_cuda_memcpy_fixed_block_size_1d_extern_shared(
            TIdx const m, TIdx const n, TIdx const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TIdx const lda,
            TElem const * const MATMUL_RESTRICT B, TIdx const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TIdx const ldc)
        {
            matmul_gemm_wrap_memcpy_host_cuda_2d(
                m, n, k,
                alpha,
                A, lda,
                B, ldb,
                beta,
                C, ldc,
                matmul_gemm_par_cuda_fixed_block_size_1d_extern_shared);
        }
    #endif
    #ifdef MATMUL_BUILD_PAR_CUDA_DYN_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        // This function only works for square blocks.
        //-----------------------------------------------------------------------------
        __global__ void matmul_gemm_par_cuda_dyn_block_size_1d_extern_shared_kernel(
            TIdx const m, TIdx const n, TIdx const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TIdx const lda,
            TElem const * const MATMUL_RESTRICT B, TIdx const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TIdx const ldc)
        {
            // Column and row of C to calculate.
            TIdx const uiGridThreadIdxX = blockIdx.x*blockDim.x + threadIdx.x;
            TIdx const uiGridThreadIdxY = blockIdx.y*blockDim.y + threadIdx.y;

            // Column and row inside the block of C to calculate.
            TIdx const uiBlockThreadIdxX = threadIdx.x;
            TIdx const uiBlockThreadIdxY = threadIdx.y;

            // The block threads extents.
            TIdx const uiBlockThreadsExtentX = blockDim.x;
            TIdx const uiBlockThreadsExtentY = blockDim.y;
            //assert(uiBlockThreadsExtentX == uiBlockThreadsExtentY);
            TIdx const & uiBlockThreadsExtent = uiBlockThreadsExtentX;

            // Shared memory used to store the current blocks of A and B.
            extern __shared__ TElem pBlockSharedA[];
            TElem * const pBlockSharedB(pBlockSharedA + uiBlockThreadsExtentX*uiBlockThreadsExtentY);

            TIdx const uiSharedBlockIdx1d(uiBlockThreadIdxY*uiBlockThreadsExtentX + uiBlockThreadIdxX);

            // If the element corresponding to the current thread is outside of the respective matrix.
            bool const bInsideA = (uiGridThreadIdxY < m);
            bool const bInsideB = (uiGridThreadIdxX < n);
            bool const bInsideC = (bInsideA && bInsideB);

            TElem dotProduct(0);

            // Loop over all blocks of A and B that are required to compute the C block.
            TIdx const uiBlockMulCount(
                static_cast<TIdx>(
                    ceil(
                        static_cast<float>(k) / static_cast<float>(uiBlockThreadsExtent))));
            for (TIdx k2(0); k2<uiBlockMulCount; ++k2)
            {
                // Copy the current blocks of A and B into shared memory in parallel.
                // If the element of the current thread is outside of the matrix, zero is written into the shared memory.
                // This is possible because zero is a result neutral extension of the matrices regarding the dot product.
                TIdx const uiAIdxX(k2*uiBlockThreadsExtentX + uiBlockThreadIdxX);
                TIdx const uiAIdx1d(uiGridThreadIdxY*lda + uiAIdxX);
                pBlockSharedA[uiSharedBlockIdx1d] =
                    ((!bInsideA) || (uiAIdxX >= k))
                    ? static_cast<TElem>(0)
                    : A[uiAIdx1d];

                TIdx const uiBIdxY(k2*uiBlockThreadsExtentY + uiBlockThreadIdxY);
                TIdx const uiBIdx1d(uiBIdxY*ldb + uiGridThreadIdxX);
                pBlockSharedB[uiSharedBlockIdx1d] =
                    ((!bInsideB) || (uiBIdxY >= k))
                    ? static_cast<TElem>(0)
                    : B[uiBIdx1d];

                // Synchronize to make sure the complete blocks are loaded before starting the computation.
                __syncthreads();

                // Compute the dot products within shared memory.
                for (TIdx k3(0); k3<uiBlockThreadsExtent; ++k3)
                {
                    dotProduct += pBlockSharedA[uiBlockThreadIdxY*uiBlockThreadsExtentX + k3]
                        * pBlockSharedB[k3*uiBlockThreadsExtentY + uiBlockThreadIdxX];
                }

                // Synchronize to make sure that the preceding computation is done before loading the next blocks of A and B.
                __syncthreads();
            }

            if (bInsideC)
            {
                TIdx const uiIdxC1d(uiGridThreadIdxY*ldc + uiGridThreadIdxX);
                C[uiIdxC1d] = alpha * dotProduct + beta * C[uiIdxC1d];
            }
        }
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        void matmul_gemm_par_cuda_dyn_block_size_1d_extern_shared(
            TIdx const m, TIdx const n, TIdx const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TIdx const lda,
            TElem const * const MATMUL_RESTRICT B, TIdx const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TIdx const ldc)
        {
            if(matmul_mat_gemm_early_out(m, n, k, alpha, beta))
            {
                return;
            }

            // MATMUL_CUDA_RT_CHECK(cudaSetDevice(0));
            cudaStream_t stream;
            MATMUL_CUDA_RT_CHECK(cudaStreamCreate(&stream));

            // Get its properties.
            cudaDeviceProp cudaDevProp;
            MATMUL_CUDA_RT_CHECK(cudaGetDeviceProperties(
                &cudaDevProp,
                0));

            TIdx vuiGridThreadExtents[] = { m, n };
            TIdx vuiBlockThreadExtents[] = { cudaDevProp.maxThreadsDim[0], cudaDevProp.maxThreadsDim[1] };

            // Restrict the max block thread extents with the grid thread extents.
            // This removes dimensions not required in the given grid thread extents.
            // This has to be done before the uiMaxBlockThreadsCount clipping to get the maximum correctly.
            for (TIdx i(0); i<2; ++i)
            {
                vuiBlockThreadExtents[i] = std::min(vuiBlockThreadExtents[i], vuiGridThreadExtents[i]);
            }

            // Restrict it to its minimum component.
            // For example (512, 256) will get (256, 256).
            auto uiMinBlockThreadExtent(vuiBlockThreadExtents[0]);
            for (TIdx i(1); i<2; ++i)
            {
                uiMinBlockThreadExtent = std::min(uiMinBlockThreadExtent, vuiBlockThreadExtents[i]);
            }
            for (TIdx i(0); i<2; ++i)
            {
                vuiBlockThreadExtents[i] = uiMinBlockThreadExtent;
            }

            // Adjust vuiBlockThreadExtents if its product is too large.
            if ((vuiBlockThreadExtents[0] * vuiBlockThreadExtents[1]) > cudaDevProp.maxThreadsPerBlock)
            {
                // Satisfy the following equation:
                // udaDevProp.maxThreadsPerBlock >= vuiBlockThreadExtents[0]*vuiBlockThreadExtents[1]
                // For example 1024 >= 512 * 512

                // For equal block thread extent this is easily the nth root of cudaDevProp.maxThreadsPerBlock.
                double const fNthRoot(std::pow(cudaDevProp.maxThreadsPerBlock, 1.0 / 2.0));
                auto const uiNthRoot(static_cast<TIdx>(fNthRoot));
                for (TIdx i(0); i<2; ++i)
                {
                    vuiBlockThreadExtents[i] = uiNthRoot;
                }
            }

            // Set the grid block extents (rounded to the next integer not less then the quotient.
            TIdx vuiGridBlockExtents[] = { 1, 1 };
            for (TIdx i(0); i<2; ++i)
            {
                vuiGridBlockExtents[i] =
                    static_cast<TIdx>(
                        std::ceil(static_cast<double>(vuiGridThreadExtents[i])
                            / static_cast<double>(vuiBlockThreadExtents[i])));
            }

            dim3 const dimBlock(vuiBlockThreadExtents[0], vuiBlockThreadExtents[1]);
            dim3 const dimGrid(vuiGridBlockExtents[0], vuiGridBlockExtents[1]);

            MATMUL_CUDA_RT_CHECK(cudaSetDevice(0));
            matmul_gemm_par_cuda_dyn_block_size_1d_extern_shared_kernel<<<
                dimGrid,
                dimBlock,
                2u*sizeof(TElem)*vuiBlockThreadExtents[0] * vuiBlockThreadExtents[1],
                stream>>>(
                    m, n, k,
                    alpha,
                    A, lda,
                    B, ldb,
                    beta,
                    C, ldc);

            // MATMUL_CUDA_RT_CHECK(cudaSetDevice(0));
            MATMUL_CUDA_RT_CHECK(cudaStreamSynchronize(stream));
            // MATMUL_CUDA_RT_CHECK(cudaSetDevice(0));
            MATMUL_CUDA_RT_CHECK(cudaStreamDestroy(stream));

            //MATMUL_CUDA_RT_CHECK(cudaDeviceSynchronize());
        }
    #endif
    #ifdef MATMUL_BUILD_PAR_CUDA_MEMCPY_DYN_BLOCK_SIZE
        //-----------------------------------------------------------------------------
        //
        //-----------------------------------------------------------------------------
        void matmul_gemm_par_cuda_memcpy_dyn_block_size_1d_extern_shared(
            TIdx const m, TIdx const n, TIdx const k,
            TElem const alpha,
            TElem const * const MATMUL_RESTRICT A, TIdx const lda,
            TElem const * const MATMUL_RESTRICT B, TIdx const ldb,
            TElem const beta,
            TElem * const MATMUL_RESTRICT C, TIdx const ldc)
        {
            matmul_gemm_wrap_memcpy_host_cuda_2d(
                m, n, k,
                alpha,
                A, lda,
                B, ldb,
                beta,
                C, ldc,
                matmul_gemm_par_cuda_dyn_block_size_1d_extern_shared);
        }
    #endif
#endif
